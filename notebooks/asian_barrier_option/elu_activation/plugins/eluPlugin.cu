#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cassert>
#include <cstring>
#include <vector>

#include "NvInfer.h"
#include "eluPlugin.h"
#include "pluginKernels.h"
#include "common.h"
#include "logger.h"

using namespace nvinfer1;

namespace elu
{

// constants for approximating the normal cdf
constexpr float A = 0.0;
constexpr float B = 1.0; // sqrt(2.0/M_PI)

template <typename T, unsigned TPB>
__global__ void eluKernel(const T a, const T b, int n, const T* input, T* output)
{

    const int idx = blockIdx.x * TPB + threadIdx.x;

    if (idx < n)
    {
        const T in = input[idx];
        const T tmp = exp(in) - b;
        const T result = (a > in ? a : in) + (a < tmp ? a : tmp);
        output[idx] = result;
    }
}

inline int computeElu(hipStream_t stream, int n, const float* input, float* output)
{

    constexpr int blockSize = 256;
    const int gridSize = (n + blockSize - 1) / blockSize;
    eluKernel<float, blockSize><<<gridSize, blockSize, 0, stream>>>(A, B, n, input, output);

    CHECK(hipPeekAtLastError());
    return 0;
}

inline int computeElu(hipStream_t stream, int n, const half* input, half* output)
{
    const int blockSize = 256;

    if (0 == (n & 1))
    {
        const int n2 = n / 2;

        const int gridSize = (n2 + blockSize - 1) / blockSize;
        const half2 A2 = __floats2half2_rn(A, A);
        const half2 B2 = __floats2half2_rn(B, B);
        const half2* input2 = reinterpret_cast<const half2*>(input);
        half2* output2 = reinterpret_cast<half2*>(output);
        eluKernel<half2, blockSize><<<gridSize, blockSize, 0, stream>>>(A2, B2, n2, input2, output2);
    }
    else
    {
        const int gridSize = (n + blockSize - 1) / blockSize;
        eluKernel<half, blockSize><<<gridSize, blockSize, 0, stream>>>(A, B, n, input, output);
    }

    CHECK(hipPeekAtLastError());
    return 0;
}

namespace
{
static const char* GELU_PLUGIN_VERSION{"1"};
static const char* GELU_PLUGIN_NAME{"CustomEluPluginDynamic"};
} // namespace

// Static class fields initialization
PluginFieldCollection EluPluginDynamicCreator::mFC{};
std::vector<PluginField> EluPluginDynamicCreator::mPluginAttributes;

REGISTER_TENSORRT_PLUGIN(EluPluginDynamicCreator);

EluPluginDynamic::EluPluginDynamic(const std::string name)
    : mLayerName(name)
{
}

EluPluginDynamic::EluPluginDynamic(const std::string name, const void* data, size_t length)
    : mLayerName(name)
{

    gLogVerbose << "Elu Deser start" << std::endl;
    const char* d = static_cast<const char*>(data);
    const char* a = d;
    mType = readFromBuffer<DataType>(d);
    assert(d == a + length);
    gLogVerbose << "Elu Deser done" << std::endl;
}
// IPluginV2DynamicExt Methods
nvinfer1::IPluginV2DynamicExt* EluPluginDynamic::clone() const
{
    return new EluPluginDynamic(mLayerName);
}

nvinfer1::DimsExprs EluPluginDynamic::getOutputDimensions(int outputIndex, const nvinfer1::DimsExprs* inputs, int nbInputs, nvinfer1::IExprBuilder& exprBuilder)
{
    return inputs[0];
}

bool EluPluginDynamic::supportsFormatCombination(int pos, const nvinfer1::PluginTensorDesc* inOut, int nbInputs, int nbOutputs)
{

    const PluginTensorDesc& input = inOut[0];
    if (pos == 0)
    {
        return (input.type == DataType::kFLOAT || input.type == DataType::kHALF)
            && (input.format == TensorFormat::kLINEAR);
    }
    if (pos == 1)
    {
        const PluginTensorDesc& output = inOut[1];
        return (input.type == output.type) && (output.format == TensorFormat::kLINEAR);
    }
    return false;
}

void EluPluginDynamic::configurePlugin(const nvinfer1::DynamicPluginTensorDesc* in, int nbInputs,
    const nvinfer1::DynamicPluginTensorDesc* out, int nbOutputs)
{
    mType = in[0].desc.type;
}

size_t EluPluginDynamic::getWorkspaceSize(const nvinfer1::PluginTensorDesc* inputs, int nbInputs,
    const nvinfer1::PluginTensorDesc* outputs, int nbOutputs) const
{
    return 0;
}
int EluPluginDynamic::enqueue(const nvinfer1::PluginTensorDesc* inputDesc,
    const nvinfer1::PluginTensorDesc* outputDesc, const void* const* inputs, void* const* outputs, void* workspace,
    hipStream_t stream)
{

    const int inputVolume = samplesCommon::volume(inputDesc[0].dims);
    int status = -1;

    // Our plugin outputs only one tensor
    // Launch CUDA kernel wrapper and save its return value
    if (mType == DataType::kFLOAT)
    {
        const float* input = static_cast<const float*>(inputs[0]);
        float* output = static_cast<float*>(outputs[0]);
        status = computeElu(stream, inputVolume, input, output);
    }
    else if (mType == DataType::kHALF)
    {
        const half* input = static_cast<const half*>(inputs[0]);
        half* output = static_cast<half*>(outputs[0]);
        status = computeElu(stream, inputVolume, input, output);
    }
    else
    {
        assert(false);
    }

    return status;
}

// IPluginV2Ext Methods
nvinfer1::DataType EluPluginDynamic::getOutputDataType(int index, const nvinfer1::DataType* inputTypes, int nbInputs) const
{
    assert(index == 0);
    assert(inputTypes[0] == DataType::kFLOAT || inputTypes[0] == DataType::kHALF);
    return inputTypes[0];
}

// IPluginV2 Methods

const char* EluPluginDynamic::getPluginType() const
{
    return GELU_PLUGIN_NAME;
}

const char* EluPluginDynamic::getPluginVersion() const
{
    return GELU_PLUGIN_VERSION;
}

int EluPluginDynamic::getNbOutputs() const
{
    return 1;
}

int EluPluginDynamic::initialize()
{
    return 0;
}

void EluPluginDynamic::terminate() {}

size_t EluPluginDynamic::getSerializationSize() const
{
    return sizeof(DataType);
}

void EluPluginDynamic::serialize(void* buffer) const
{
    char *d = static_cast<char*>(buffer), *a = d;
    writeToBuffer(d, mType);
    assert(d == a + getSerializationSize());
}

void EluPluginDynamic::destroy()
{
    // This gets called when the network containing plugin is destroyed
    delete this;
}

void EluPluginDynamic::setPluginNamespace(const char* libNamespace)
{
    mNamespace = libNamespace;
}

const char* EluPluginDynamic::getPluginNamespace() const
{
    return mNamespace.c_str();
}

///////////////

EluPluginDynamicCreator::EluPluginDynamicCreator()
{

    // Fill PluginFieldCollection with PluginField arguments metadata
    mFC.nbFields = mPluginAttributes.size();
    mFC.fields = mPluginAttributes.data();
}

const char* EluPluginDynamicCreator::getPluginName() const
{
    return GELU_PLUGIN_NAME;
}

const char* EluPluginDynamicCreator::getPluginVersion() const
{
    return GELU_PLUGIN_VERSION;
}

const PluginFieldCollection* EluPluginDynamicCreator::getFieldNames()
{
    return &mFC;
}

IPluginV2* EluPluginDynamicCreator::createPlugin(const char* name, const PluginFieldCollection* fc)
{
    gLogVerbose << "Creating EluPluginDynamic...\n";
    EluPluginDynamic* p = new EluPluginDynamic(name);
    return p;
}

IPluginV2* EluPluginDynamicCreator::deserializePlugin(const char* name, const void* serialData, size_t serialLength)
{
    // This object will be deleted when the network is destroyed, which will
    // call EluPluginDynamic::destroy()
    return new EluPluginDynamic(name, serialData, serialLength);
}

void EluPluginDynamicCreator::setPluginNamespace(const char* libNamespace)
{
    mNamespace = libNamespace;
}

const char* EluPluginDynamicCreator::getPluginNamespace() const
{
    return mNamespace.c_str();
}
}
