#include "hip/hip_runtime.h"
#include <vector>
#include <stdio.h>
#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand.h>
 
#define CHECKCURAND(expression)                         \
  {                                                     \
    hiprandStatus_t status = (expression);                         \
    if (status != HIPRAND_STATUS_SUCCESS) {                        \
      std::cerr << "Curand Error on line " << __LINE__<< std::endl;     \
      std::exit(EXIT_FAILURE);                                          \
    }                                                                   \
  }

// atomicAdd is introduced for compute capability >=6.0
#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
#else
__device__ double atomicAdd(double* address, double val)
{
      printf("device arch <=600\n");
        unsigned long long int* address_as_ull = (unsigned long long int*)address;
          unsigned long long int old = *address_as_ull, assumed;
            do {
                    assumed = old;
                        old = atomicCAS(address_as_ull, assumed,
                                                    __double_as_longlong(val + __longlong_as_double(assumed)));
                          } while (assumed != old);
              return __longlong_as_double(old);
}
#endif

__global__ void sumPayoffKernel(float *d_s, const unsigned N_PATHS, double *mysum)
{
  unsigned idx =  threadIdx.x + blockIdx.x * blockDim.x;
  unsigned stride = blockDim.x * gridDim.x;
  unsigned tid = threadIdx.x;

  extern __shared__ double smdata[];
  smdata[tid] = 0.0;

  for (unsigned i = idx; i<N_PATHS; i+=stride)
  {
    smdata[tid] += (double) d_s[i];
  }

  for (unsigned s=blockDim.x/2; s>0; s>>=1)
  {
    __syncthreads();
    if (tid < s) smdata[tid] += smdata[tid + s];
  }

  if (tid == 0)
  {
    atomicAdd(mysum, smdata[0]);
  }
}

__global__ void barrier_option(
    float *d_s,
    const float T,
    const float K,
    const float B,
    const float S0,
    const float sigma,
    const float mu,
    const float r,
    const float * d_normals,
    const long N_STEPS,
    const long N_PATHS)
{
  unsigned idx =  threadIdx.x + blockIdx.x * blockDim.x;
  unsigned stride = blockDim.x * gridDim.x;
  const float tmp1 = mu*T/N_STEPS;
  const float tmp2 = exp(-r*T);
  const float tmp3 = sqrt(T/N_STEPS);
  double running_average = 0.0;

  for (unsigned i = idx; i<N_PATHS; i+=stride)
  {
    float s_curr = S0;
    for(unsigned n = 0; n < N_STEPS; n++){
       s_curr += tmp1 * s_curr + sigma*s_curr*tmp3*d_normals[i + n * N_PATHS];
       running_average += (s_curr - running_average) / (n + 1.0) ;
       if (running_average <= B){
           break;
       }
    }

    float payoff = (running_average>K ? running_average-K : 0.f);
    d_s[i] = tmp2 * payoff;
  }
}

int main(int argc, char *argv[]) {
  try {
    // declare variables and constants
    size_t N_PATHS = 8192000;
    size_t N_STEPS = 365;
    if (argc >= 2)  N_PATHS = atoi(argv[1]);

    if (argc >= 3)  N_STEPS = atoi(argv[2]);

    const float T = 1.0f;
    const float K = 110.0f;
    const float B = 100.0f;
    const float S0 = 120.0f;
    const float sigma = 0.35f;
    const float mu = 0.1f;
    const float r = 0.05f;


    double gpu_sum{0.0};

    int devID{0};
    hipDeviceProp_t deviceProps;

    checkCudaErrors(hipGetDeviceProperties(&deviceProps, devID));
    printf("CUDA device [%s]\n", deviceProps.name);
    printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProps.name, deviceProps.major, deviceProps.minor);
    // Generate random numbers on the device
    hiprandGenerator_t curandGenerator;
    CHECKCURAND(hiprandCreateGenerator(&curandGenerator, HIPRAND_RNG_PSEUDO_MTGP32));
    CHECKCURAND(hiprandSetPseudoRandomGeneratorSeed(curandGenerator, 1234ULL)) ;

    const size_t N_NORMALS = (size_t)N_STEPS * N_PATHS;
    float *d_normals;
    checkCudaErrors(hipMalloc(&d_normals, N_NORMALS * sizeof(float)));
    CHECKCURAND(hiprandGenerateNormal(curandGenerator, d_normals, N_NORMALS, 0.0f, 1.0f));
    hipDeviceSynchronize();

  	// before kernel launch, check the max potential blockSize
  	int BLOCK_SIZE, GRID_SIZE;
  	checkCudaErrors(hipOccupancyMaxPotentialBlockSize(&GRID_SIZE,
  	                                                   &BLOCK_SIZE,
  	                                                   barrier_option,
  	                                                   0, N_PATHS));

  	std::cout << "suggested block size " << BLOCK_SIZE
  	          << " \nsuggested grid size " << GRID_SIZE
  	          << std::endl;

  	std::cout << "Used grid size " << GRID_SIZE << std::endl;

  	// Kernel launch
  	auto t1=std::chrono::high_resolution_clock::now();

  	float *d_s;
  	checkCudaErrors(hipMalloc(&d_s, N_PATHS*sizeof(float)));

  	auto t3=std::chrono::high_resolution_clock::now();
  	barrier_option<<<GRID_SIZE, BLOCK_SIZE>>>(d_s, T, K, B, S0, sigma, mu, r, d_normals, N_STEPS, N_PATHS);
  	hipDeviceSynchronize();
  	auto t4=std::chrono::high_resolution_clock::now();

  	double* mySum;
  	checkCudaErrors(hipMallocManaged(&mySum, sizeof(double)));
  	sumPayoffKernel<<<GRID_SIZE, BLOCK_SIZE, BLOCK_SIZE*sizeof(double)>>>(d_s, N_PATHS, mySum);
  	hipDeviceSynchronize();
  	auto t5=std::chrono::high_resolution_clock::now();

  	std::cout << "sumPayoffKernel takes "
  	          << std::chrono::duration_cast<std::chrono::microseconds>(t5-t4).count() / 1000.f
  	          << " ms\n";

  	gpu_sum = mySum[0] / N_PATHS;

  	auto t2=std::chrono::high_resolution_clock::now();

  	// clean up
  	CHECKCURAND(hiprandDestroyGenerator( curandGenerator )) ;
  	checkCudaErrors(hipFree(d_s));
  	checkCudaErrors(hipFree(d_normals));
  	checkCudaErrors(hipFree(mySum));

  	std::cout << "price "
              << gpu_sum
              << " time "
  	          << std::chrono::duration_cast<std::chrono::microseconds>(t5-t1).count() / 1000.f
  	          << " ms\n";
  }

  catch(std::
        exception& e)
  {
    std::cout<< "exception: " << e.what() << "\n";
  }
}
